#include "hip/hip_runtime.h"
#include "post_process.cuh"
namespace ai
{
    namespace postprocess
    {
        // keepflag主要是用来进行nms时候判断是否将该框抛弃
        const int NUM_BOX_ELEMENT = 7; // left, top, right, bottom, confidence, class, keepflag
        static __device__ void affine_project(float *matrix, float x, float y, float *ox, float *oy)
        {
            *ox = matrix[0] * x + matrix[1] * y + matrix[2];
            *oy = matrix[3] * x + matrix[4] * y + matrix[5];
        }

        static __global__ void decode_kernel_common(float *predict, int num_bboxes, int num_classes,
                                                    int output_cdim, float confidence_threshold,
                                                    float *invert_affine_matrix, float *parray,
                                                    int MAX_IMAGE_BOXES)
        {
            int position = blockDim.x * blockIdx.x + threadIdx.x;
            if (position >= num_bboxes)
                return;

            // pitem就获取了每个box的首地址
            // output_cdim是指每个box中有几个元素，可以根据onnx最后的输出定
            // 每个box的元素一般是(根据你的模型可修改,objectness是yolo系列的是否有物体得分)：left,top,right,bottom,objectness,class0,class1,...,classn
            float *pitem = predict + output_cdim * position;
            float objectness = pitem[4];
            if (objectness < confidence_threshold)
                return;

            // 从多个类别得分中，找出最大类别的class_score+label
            float *class_confidence = pitem + 5;
            float confidence = *class_confidence++; // 取class1给confidence并且class_confidence自增1
            int label = 0;
            // ++class_confidence和class_confidence++在循环中执行的结果是一样的，都是执行完循环主体后再加一
            for (int i = 1; i < num_classes; ++i, ++class_confidence)
            {
                if (*class_confidence > confidence)
                {
                    confidence = *class_confidence;
                    label = i;
                }
            }

            confidence *= objectness; // yolo系列的最终得分是两者相乘
            if (confidence < confidence_threshold)
                return;

            // cuda的原子操作：int atomicAdd(int *M,int V); 它们把一个内存位置M和一个数值V作为输入。
            // 与原子函数相关的操作在V上执行，数值V早已存储在内存地址*M中了，然后将相加的结果写到同样的内存位置中。
            int index = atomicAdd(parray, 1); // 所以这段代码意思是用parray[0]来计算boxes的总个数
            if (index >= MAX_IMAGE_BOXES)
                return;

            float cx = *pitem++;
            float cy = *pitem++;
            float width = *pitem++;
            float height = *pitem++;
            float left = cx - width * 0.5f;
            float top = cy - height * 0.5f;
            float right = cx + width * 0.5f;
            float bottom = cy + height * 0.5f;
            // boxes映射回相对于真实图片的尺寸
            affine_project(invert_affine_matrix, left, top, &left, &top);
            affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

            // parray+1之后的值全部用来存储boxes元素，每个框有NUM_BOX_ELEMENT个元素
            float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
            *pout_item++ = left;
            *pout_item++ = top;
            *pout_item++ = right;
            *pout_item++ = bottom;
            *pout_item++ = confidence;
            *pout_item++ = label;
            *pout_item++ = 1; // 1 = keep, 0 = ignore
        }

        static __device__ float box_iou(
            float aleft, float atop, float aright, float abottom,
            float bleft, float btop, float bright, float bbottom)
        {

            float cleft = max(aleft, bleft);
            float ctop = max(atop, btop);
            float cright = min(aright, bright);
            float cbottom = min(abottom, bbottom);

            float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
            if (c_area == 0.0f)
                return 0.0f;

            float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
            float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
            return c_area / (a_area + b_area - c_area);
        }

        static __global__ void nms_kernel(float *bboxes, int max_objects, float threshold)
        {

            int position = (blockDim.x * blockIdx.x + threadIdx.x);
            int count = min((int)*bboxes, max_objects);
            if (position >= count)
                return;

            // left, top, right, bottom, confidence, class, keepflag
            float *pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
            for (int i = 0; i < count; ++i)
            {
                float *pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
                if (i == position || pcurrent[5] != pitem[5])
                    continue;

                if (pitem[4] >= pcurrent[4])
                {
                    if (pitem[4] == pcurrent[4] && i < position)
                        continue;

                    float iou = box_iou(
                        pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                        pitem[0], pitem[1], pitem[2], pitem[3]);

                    if (iou > threshold)
                    {
                        pcurrent[6] = 0; // 1=keep, 0=ignore
                        return;
                    }
                }
            }
        }

        static __global__ void decode_kernel_v8_trans(float *predict, int num_bboxes, int num_classes,
                                                      int output_cdim, float confidence_threshold,
                                                      float *invert_affine_matrix, float *parray,
                                                      int MAX_IMAGE_BOXES)
        {
            int position = blockDim.x * blockIdx.x + threadIdx.x;
            if (position >= num_bboxes)
                return;

            // yolov8和其他yolo系列的box不一样，是：left,top,right,bottom,class0,class1,...,classn
            // 然后在class0,class1,...,classn中取最大的座位score和label，去除了objectness，且是列排序，所以，需要对前面的解析代码稍微改变
            // float *pitem = predict + output_cdim * position;
            float max_confidence = *(predict + 4 * num_bboxes + position);
            int label = 0;
            for (int i = 1; i < num_classes; ++i)
            {
                if (*(predict + (4 + i) * num_bboxes + position) > max_confidence)
                {
                    max_confidence = *(predict + (4 + i) * num_bboxes + position);
                    label = i;
                }
            }
            if (max_confidence < confidence_threshold)
                return;

            int index = atomicAdd(parray, 1);
            if (index >= MAX_IMAGE_BOXES)
                return;

            float cx = *(predict + 0 * num_bboxes + position);
            float cy = *(predict + 1 * num_bboxes + position);
            float width = *(predict + 2 * num_bboxes + position);
            float height = *(predict + 3 * num_bboxes + position);
            float left = cx - width * 0.5f;
            float top = cy - height * 0.5f;
            float right = cx + width * 0.5f;
            float bottom = cy + height * 0.5f;
            affine_project(invert_affine_matrix, left, top, &left, &top);
            affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

            float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
            *pout_item++ = left;
            *pout_item++ = top;
            *pout_item++ = right;
            *pout_item++ = bottom;
            *pout_item++ = max_confidence;
            *pout_item++ = label;
            *pout_item++ = 1; // 1 = keep, 0 = ignore
        }

        void decode_detect_kernel_invoker(float *predict, int num_bboxes, int num_classes, int output_cdim,
                                          float confidence_threshold, float *invert_affine_matrix,
                                          float *parray, int MAX_IMAGE_BOXES, hipStream_t stream)
        {
            auto grid = CUDATools::grid_dims(num_bboxes);
            auto block = CUDATools::block_dims(num_bboxes);

            checkCudaKernel(decode_kernel_common<<<grid, block, 0, stream>>>(
                predict, num_bboxes, num_classes, output_cdim, confidence_threshold, invert_affine_matrix,
                parray, MAX_IMAGE_BOXES));
        }

        void nms_kernel_invoker(float *parray, float nms_threshold, int max_objects, hipStream_t stream)
        {

            auto grid = CUDATools::grid_dims(max_objects);
            auto block = CUDATools::block_dims(max_objects);
            checkCudaKernel(nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold));
        }

        void decode_detect_yolov8_kernel_invoker(float *predict, int num_bboxes, int num_classes, int output_cdim,
                                                 float confidence_threshold, float *invert_affine_matrix,
                                                 float *parray, int MAX_IMAGE_BOXES, hipStream_t stream)
        {
            auto grid = CUDATools::grid_dims(num_bboxes);
            auto block = CUDATools::block_dims(num_bboxes);
            // yolov3/v5/v7/yolox等模型的输出格式是[batch,num_boxes,output_cdim],这样，每个框的所有值是连续排列的[行排序]，方便使用
            // 但是yolov8的输出是[batch,output_cdim,num_boxes],这就超难受了，每个框的所以值都不连续，冲突是最大的[列排序]，解决方案:
            // 1. 从onnx的导出上解决，直接将其维度[batch,6,8400]-->[batch,8400,6],这样再生成engine就可以了，这个速度较快
            // 2. 从解析结果层面解决，但这会造成kernel函数执行线程存储体的冲突且冲突是最大的，所以这个方法速度稍慢，本节用这个
            checkCudaKernel(decode_kernel_v8_trans<<<grid, block, 0, stream>>>(
                predict, num_bboxes, num_classes, output_cdim, confidence_threshold, invert_affine_matrix,
                parray, MAX_IMAGE_BOXES));
        }
    }
}